/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc -arch=sm_11 bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 * credits: https://gist.github.com/mre/1392067 
 */
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <fstream>
#include <vector>
#include <iostream>
#include <algorithm>
#include <cmath>
#include <numeric>

// Helper function to check if an array is sorted
bool isSorted(const int* array, int size) {
    for (int i = 1; i < size; i++) {
        if (array[i] < array[i-1]) {
            return false;
        }
    }
    return true;
}

__global__ void bitonic_sort_step(int *dev_values, int j, int k)
{
    /* Sorting partners: i and ixj */
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i)
    {
        if ((i & k) == 0)
        {
            /* Sort ascending */
            if (dev_values[i] > dev_values[ixj])
            {
                /* exchange(i,ixj); */
                int temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i & k) != 0)
        {
            /* Sort descending */
            if (dev_values[i] < dev_values[ixj])
            {
                /* exchange(i,ixj); */
                int temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

// Helper function to get the next power of 2
int nextPowerOf2(int n)
{
    return pow(2, ceil(log2(n)));
}

/**
 * Inplace bitonic sort using CUDA.
 */
void bitonic_sort(int *values, int num_vals)
{
    // Check if num_vals is a power of 2, if not, pad the array
    int orig_num_vals = num_vals;
    int padded_size = nextPowerOf2(num_vals);
    
    // Create a padded array if needed
    int *padded_values = values;
    if (padded_size > num_vals) {
        padded_values = new int[padded_size];
        
        // Copy original values
        for (int i = 0; i < num_vals; i++) {
            padded_values[i] = values[i];
        }
        
        // Fill the rest with INT_MAX (will sort to the end)
        for (int i = num_vals; i < padded_size; i++) {
            padded_values[i] = INT_MAX;
        }
        
        num_vals = padded_size;
    }
    
    int *dev_values;
    size_t size = num_vals * sizeof(int);

    hipMalloc((void **)&dev_values, size);
    hipMemcpy(dev_values, padded_values, size, hipMemcpyHostToDevice);

    const int threads_per_block = 1024; // adjust to whatever
    int blocks = (num_vals + threads_per_block - 1) / threads_per_block;

    int j, k;
    /* Major step */
    for (k = 2; k <= num_vals; k <<= 1)
    {
        /* Minor step */
        for (j = k >> 1; j > 0; j = j >> 1)
        {
            dim3 gridDim(blocks);
            dim3 blockDim(threads_per_block);
            void *args[] = {&dev_values, &j, &k};
            hipLaunchKernel(reinterpret_cast<const void*>((void*)bitonic_sort_step), gridDim, blockDim, args, 0, NULL);
            hipDeviceSynchronize(); // Make sure each kernel completes before next launch
        }
    }
    
    // Copy back the sorted array
    if (padded_size > orig_num_vals) {
        // Copy only the original number of elements back to values
        hipMemcpy(padded_values, dev_values, size, hipMemcpyDeviceToHost);
        for (int i = 0; i < orig_num_vals; i++) {
            values[i] = padded_values[i];
        }
        delete[] padded_values;
    } else {
        hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    }
    
    hipFree(dev_values);
}

int main()
{
    // Constants for benchmarking
    const int WARMUP_RUNS = 5;
    const int BENCHMARK_RUNS = 10000;
    
    // Read integers from file
    std::vector<int> original_values;
    std::ifstream file("random_array_of_10^4_integers.txt");
    
    if (!file.is_open()) {
        std::cerr << "Error: Could not open array.txt" << std::endl;
        return 1;
    }
    
    int num;
    while (file >> num) {
        original_values.push_back(num);
    }
    file.close();
    
    int num_vals = original_values.size();
    std::cout << "Array size: " << num_vals << " elements" << std::endl;
    
    // Create array from vector for original values
    int* original_array = new int[num_vals];
    for (int i = 0; i < num_vals; i++) {
        original_array[i] = original_values[i];
    }
    
    // Vector to store all benchmark run times
    std::vector<int> run_times;
    
    // Warm-up runs
    std::cout << "Performing " << WARMUP_RUNS << " warm-up runs..." << std::endl;
    for (int i = 0; i < WARMUP_RUNS; i++) {
        // Create a copy of the original array for each run
        int* array_copy = new int[num_vals];
        for (int j = 0; j < num_vals; j++) {
            array_copy[j] = original_array[j];
        }
        
        // Sort and ignore timing
        bitonic_sort(array_copy, num_vals);
        
        // Check if array is sorted after first warm-up run only
        if (i == 0) {
            bool sorted = isSorted(array_copy, num_vals);
            if (!sorted) {
                std::cerr << "ERROR: Array not correctly sorted after first warm-up run!" << std::endl;
                delete[] array_copy;
                delete[] original_array;
                return 1;  // Exit with error code
            }
            std::cout << "Array correctly sorted, continuing with benchmark." << std::endl;
        }
        
        delete[] array_copy;
    }
    
    // Benchmark runs
    std::cout << "\nPerforming " << BENCHMARK_RUNS << " benchmark runs..." << std::endl;
    for (int i = 0; i < BENCHMARK_RUNS; i++) {
        // Create a copy of the original array for each run
        int* array_copy = new int[num_vals];
        for (int j = 0; j < num_vals; j++) {
            array_copy[j] = original_array[j];
        }
        
        // Create CUDA events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        // Start timing
        hipEventRecord(start);
        
        // Sort
        bitonic_sort(array_copy, num_vals);
        
        // Stop timing
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        // Calculate elapsed time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        // Convert to integer microseconds
        int microseconds = static_cast<int>(milliseconds * 1000.0f);
        
        // Store the duration
        run_times.push_back(microseconds);
        
        // Clean up
        delete[] array_copy;
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    
    // Calculate statistics
    // 1. Mean
    double mean = std::accumulate(run_times.begin(), run_times.end(), 0.0) / run_times.size();
    
    // 2. Median (sort the vector first)
    std::vector<int> sorted_times = run_times;
    std::sort(sorted_times.begin(), sorted_times.end());
    double median = sorted_times.size() % 2 == 0 
                  ? (sorted_times[sorted_times.size()/2 - 1] + sorted_times[sorted_times.size()/2]) / 2.0
                  : sorted_times[sorted_times.size()/2];
    
    // 3. Standard Deviation
    double variance = 0.0;
    for (const auto& time : run_times) {
        variance += std::pow(time - mean, 2);
    }
    variance /= run_times.size();
    double std_dev = std::sqrt(variance);
    
    // Output statistics
    std::cout << "\nStatistics for " << BENCHMARK_RUNS << " runs (in microseconds):" << std::endl;
    std::cout << "Mean: " << mean << std::endl;
    std::cout << "Median: " << median << std::endl;
    std::cout << "Standard Deviation: " << std_dev << std::endl;
    
    // Clean up
    delete[] original_array;
    
    return 0;
}

